/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>
#include <stdexcept>

#define CHECK_TENSOR_TYPE(tensor, expected_type_enum, expected_type_str) \
	do { \
		if ((tensor).scalar_type() != (expected_type_enum)) { \
			throw std::runtime_error( \
				"Expected tensor type: " + std::string(expected_type_str) + \
				", but got: " + at::toString((tensor).scalar_type()) \
			); \
		} \
	} while (0)

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug,
	const float blur_radius,
	const torch::Tensor& boundary_points_3d,
	const torch::Tensor& boundary_points_bprimitive_id,
	const torch::Tensor& bprimitive_image,
	const torch::Tensor& gaussian_bprimitive_id)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }

  CHECK_TENSOR_TYPE(bprimitive_image, torch::kInt32, "int32");
  CHECK_TENSOR_TYPE(gaussian_bprimitive_id, torch::kInt32, "int32");
  CHECK_TENSOR_TYPE(boundary_points_bprimitive_id, torch::kInt32, "int32");
  
  const int P = means3D.size(0);
  const int PB = boundary_points_3d.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  torch::Tensor boundaryBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binning_boundaryBuffer = torch::empty({0}, options.device(device));
  torch::Tensor img_boundaryBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  std::function<char*(size_t)> boundaryFunc = resizeFunctional(boundaryBuffer);
  std::function<char*(size_t)> binning_boundaryFunc = resizeFunctional(binning_boundaryBuffer);
  std::function<char*(size_t)> img_boundaryFunc = resizeFunctional(img_boundaryBuffer);
  
  int rendered = 0;
  int boundary_rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  auto&& [rendered, boundary_rendered] = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
		boundaryFunc,
		binning_boundaryFunc,
		img_boundaryFunc,
		P, degree, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug,
		PB, blur_radius,
		boundary_points_3d.contiguous().data<float>(),
		boundary_points_bprimitive_id.contiguous().data<int>(),
		bprimitive_image.contiguous().data<int>(),
		gaussian_bprimitive_id.contiguous().data<int>());
  }
  return std::make_tuple(rendered, boundary_rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer, boundaryBuffer, binning_boundaryBuffer, img_boundaryBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const int RB,
	const float blur_radius,
	const torch::Tensor& boundary_points_3d,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const torch::Tensor& boundaryBuffer,
	const torch::Tensor& binning_boundaryBuffer,
	const torch::Tensor& image_boundaryBuffer,
    const torch::Tensor& bprimitive_image,
    const torch::Tensor& gaussian_bprimitive_id,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int PB = boundary_points_3d.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_dmeans3D_boundary = torch::zeros({PB, 3}, means3D.options());
  torch::Tensor dL_dmeans2D_boundary = torch::zeros({PB, 2}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  PB, blur_radius, RB,
	  boundary_points_3d.contiguous().data<float>(),
	  reinterpret_cast<char*>(boundaryBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binning_boundaryBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(image_boundaryBuffer.contiguous().data_ptr()),
	  bprimitive_image.contiguous().data<int>(),
	  gaussian_bprimitive_id.contiguous().data<int>(),
	  dL_dmeans3D_boundary.contiguous().data<float>(),
	  dL_dmeans2D_boundary.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations, dL_dmeans3D_boundary);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}